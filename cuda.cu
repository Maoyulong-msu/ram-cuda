
#include <hip/hip_runtime.h>
#include<iostream>
#include<hip/hip_runtime_api.h>
#include<time.h>
#include<stdlib.h>

#define SAFE_CALL(CallInstruction){ \
	hipError_t cuerr=CallInstruction; \
	if(cuerr!=hipSuccess){ \
		printf("CUDA error:%s at call \"" #CallInstruction"\"\n",hipGetErrorString(cuerr));\
			throw "error in CUDA API function,aborting...";\
	} \
}

#define SAFE_KERNEL_CALL(KernelCallInstruction){\
	KernelCallInstruction; \
	hipError_t cuerr=hipGetLastError();\
	if(cuerr!=hipSuccess){\
		printf("CUDA error in kernel launch:%s at kernel \"" #KernelCallInstruction "\"\n",hipGetErrorString(cuerr)); \
			throw "error in CUDA kernel launch,aborting...";\
	}\
	cuerr=hipDeviceSynchronize();\
	if(cuerr!=hipSuccess){\
		printf("CUDA error in kernel execution:%s at kernel\"" #KernelCallInstruction "\"\n",hipGetErrorString(cuerr));\
			throw "error in CUDA kernel execution,aborting...";\
	}\
}

#define size_mb 1048576

//ядро
__global__ void ram(int *ptrs,int *result,int *data,int num)
{
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx<num)
	{
		result[idx]=data[ptrs[idx]];
	}
}


int main(int argc,char **argv)
{
	int n=atoi(argv[1]);
	long int cached_array_size=size_mb*n;
	long int large_array_size=size_mb*1024;

	//host data
	int *h_ptrs,*h_result,*h_data;

	h_ptrs=(int*)malloc(large_array_size);	
	h_result=(int*)malloc(large_array_size);
	h_data=(int*)malloc(cached_array_size);

	for(int i=0;i<cached_array_size;i++)
	{
		h_data[i]=i;
	}

	for(int i=0;i<large_array_size;i++)
	{
		h_result[i]=0;
		h_ptrs[i]=rand()%cached_array_size;
	}

	//device data
	int *d_ptrs,*d_result,*d_data;

	SAFE_CALL(hipMalloc(&d_ptrs,large_array_size));
	SAFE_CALL(hipMalloc(&d_result,large_array_size));
	SAFE_CALL(hipMalloc(&d_data,cached_array_size));

	SAFE_CALL(hipMemcpy(d_ptrs,h_ptrs,large_array_size,hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(d_result,h_result,large_array_size,hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(d_data,h_data,cached_array_size,hipMemcpyHostToDevice));
	int blocksize=1024;
	int gridsize=(((large_array_size/sizeof(int))-1)/blocksize)+1;

	hipEvent_t start,finish;
	SAFE_CALL(hipEventCreate(&start));
	SAFE_CALL(hipEventCreate(&finish));
	
	SAFE_CALL(hipDeviceSynchronize());

	SAFE_CALL(hipEventRecord(start));
	SAFE_KERNEL_CALL((ram<<<gridsize,blocksize>>>(d_ptrs,d_result,d_data,large_array_size/sizeof(int))));
	SAFE_CALL(hipEventRecord(finish));

	SAFE_CALL(hipMemcpy(h_result,d_result,large_array_size,hipMemcpyDeviceToHost));
	SAFE_CALL(hipEventSynchronize(finish));
	
	float milliseconds=0;
	SAFE_CALL(hipEventElapsedTime(&milliseconds,start,finish));
	
	printf("Used time is(s):%f",milliseconds/1000);
	printf("Effective Bandwidth(GB/s):%fn",large_array_size*3/milliseconds/1e6);
	
	FILE *bandwidth=fopen("bandwidth.dat","a");
	fprintf(bandwidth,"%d %fn",n,large_array_size*3/milliseconds/1e6);

	SAFE_CALL(hipFree(d_ptrs));
	SAFE_CALL(hipFree(d_result));
	SAFE_CALL(hipFree(d_data));

	free(h_ptrs);
	free(h_result);
	free(h_data);

	return 0;
}
